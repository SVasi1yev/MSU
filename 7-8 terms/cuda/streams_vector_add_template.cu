#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void sample_vec_add(int size = 1048576)
{
    printf("sample\n");

    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    float *test;
    
    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipHostMalloc(&test, nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        test[i] = a[i] + b[i];
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    //printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    //printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

    hipMemcpy(c, c_d, nBytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();

    for(int i = 0; i < 20; i++)
    {
        printf("%f\n", c[i]);
    }
    for(int i = 0; i < n; i++)
    {
        if(c[i] != test[i])
        {
            printf("INCORRECT!\n");
            break;
        }
    }
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(test);
}

void streams_vec_add(int size = 1048576)
{
    printf("streams\n");

    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    float *test;
    
    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipHostMalloc(&test, nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)n/(2 * block.x) + 1);
    
    for(int i=0;i<n;i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        test[i] = a[i] + b[i];
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d, nBytes);
    hipMalloc((void **)&b_d, nBytes);
    hipMalloc((void **)&c_d, nBytes);

    float *a_offset = a + n / 2, *b_offset = b + n / 2, *c_offset = c + n / 2;
    float *a_d_offset = a_d + n / 2, *b_d_offset = b_d + n / 2, *c_d_offset = c_d + n / 2;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    // printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpyAsync(a_d, a, nBytes / 2, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(b_d, b, nBytes / 2, hipMemcpyHostToDevice, stream1);

    hipMemcpyAsync(a_d_offset, a_offset, nBytes / 2, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(b_d_offset, b_offset, nBytes / 2, hipMemcpyHostToDevice, stream2);
    
    // printf("Doing GPU Vector add\n");
    vectorAddGPU<<<grid, block, 0, stream1>>>(a_d, b_d, c_d, n / 2);
    vectorAddGPU<<<grid, block, 0, stream2>>>(a_d_offset, b_d_offset, c_d_offset, n / 2);

    hipMemcpyAsync(c, c_d, nBytes / 2, hipMemcpyDeviceToHost, stream1);

    hipMemcpyAsync(c_offset, c_d_offset, nBytes / 2, hipMemcpyDeviceToHost, stream2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();

    for(int i = 0; i < 20; i++)
    {
        printf("%f\n", c[i]);
    }
    for(int i = 0; i < n; i++)
    {
        if(c[i] != test[i])
        {
            printf("INCORRECT!\n");
            break;
        }
    }
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(test);
}


int main(int argc, char **argv)
{
    sample_vec_add(atoi(argv[1]));
    streams_vec_add(atoi(argv[1]));

    return 0;
}
