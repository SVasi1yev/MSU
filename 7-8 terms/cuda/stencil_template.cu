// UCSC CMPE220 Advanced Parallel Processing 
// Prof. Heiner Leitz
// Author: Marcelo Siero.
// Modified from code by:: Andreas Goetz (agoetz@sdsc.edu)
// CUDA program to perform 1D stencil operation in parallel on the GPU
//
// /* FIXME */ COMMENTS ThAT REQUIRE ATTENTION


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cuda_device_runtime_api.h>

// define vector length, stencil radius, 
#define N (1024*1024*512l)
#define RADIUS 3
#define GRIDSIZE 524288
#define BLOCKSIZE 1024

int gridSize  = GRIDSIZE;
int blockSize = BLOCKSIZE;

hipEvent_t start, stop;
float gpu_elapsed_time_ms;


void cudaErrorCheck() {
   // FIXME: Add code that finds the last error for CUDA functions performed.
   // Upon getting an error have it print out a meaningful error message as 
   //  provided by the CUDA API, then exit with an error exit code.
}

void start_timer() {
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
}

float stop_timer() {
   // FIXME: ADD TIMING CODE, HERE, USE GLOBAL VARIABLES AS NEEDED.
   hipDeviceSynchronize();
   // time counting terminate
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
   return gpu_elapsed_time_ms;
}

hipDeviceProp_t prop;
void getDeviceProperties() {
   // FIXME: Implement this function so as to acquire and print the following 
   // device properties:
   //    Major and minor CUDA capability, total device global memory,
   //    size of shared memory per block, number of registers per block,
   //    warp size, max number of threads per block, number of multi-prccessors
   //    (SMs) per device, Maximum number of threads per block dimension (x,y,z),
   //    Maximumum number of blocks per grid dimension (x,y,z).
   //
   // These properties can be useful to dynamically optimize programs.  For
   // instance the number of SMs can be useful as a heuristic to determine
   // how many is a good number of blocks to use.  The total device global
   // memory might be important to know just how much data to operate on at
   // once.
}

void newline() { std::cout << std::endl; };

void printThreadSizes() {
   int noOfThreads = gridSize * blockSize;
   printf("Blocks            = %d\n", gridSize);  // no. of blocks to launch.
   printf("Threads per block = %d\n", blockSize); // no. of threads to launch.
   printf("Total threads     = %d\n", noOfThreads);
   printf("Number of grids   = %d\n", (N + noOfThreads -1)/ noOfThreads);
}

// -------------------------------------------------------
// CUDA device function that performs 1D stencil operation
// -------------------------------------------------------
__global__ void stencil_1D(int *in, int *out, long dim){
  int it = threadIdx.x;

  // long index = threadIdx.x + blockDim.x * blockIdx.x;
  long gindex = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  
  __shared__ int array[2 * RADIUS + BLOCKSIZE];
  
  // Go through all data
  // Step all threads in a block to avoid synchronization problem
   array[it + RADIUS] = in[gindex];
   if(it < RADIUS)
   {
      if(gindex < RADIUS)
      {
         array[it] = 0;
      }
      else
      {
         array[it] = in[gindex - RADIUS];
      }
   } 
   else if(it > (BLOCKSIZE - RADIUS - 1))
   {
      if(gindex > N - RADIUS - 1)
      {
         array[it + 2 * RADIUS] = 0;
      }
      else
      {
         array[it + 2 * RADIUS] = in[gindex + RADIUS];
      }
   }
   __syncthreads();

    /* FIXME PART 2 - MODIFIY PROGRAM TO USE SHARED MEMORY. */

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
	        result += array[it + RADIUS + offset];
    }

    // Store the result
    if (gindex < dim)
      out[gindex] = result;

    // Update global index and quit if we are done
    __syncthreads();
}

#define True  1
#define False 0
void checkResults(int *h_in, int *h_out, int DoCheck=True) {
   // DO NOT CHANGE THIS CODE.
   // CPU calculates the stencil from data in *h_in
   // if DoCheck is True (default) it compares it with *h_out
   // to check the operation of this code.
   // If DoCheck is set to False, it can be used to time the CPU.
   int i, j, ij, result, err;
   err = 0;
   for (i=0; i<N; i++){  // major index.
      result = 0;
      for (j=-RADIUS; j<=RADIUS; j++){
         ij = i+j;
         if (ij>=0 && ij<N)
            result += h_in[ij];
      }
      if (DoCheck) {  // print out some errors for debugging purposes.
         if (h_out[i] != result) { // count errors.
            err++;
            if (err < 8) { // help debug
               printf("h_out[%d]=%d should be %d\n",i,h_out[i], result);
            };
         }
      } else {  // for timing purposes.
         h_out[i] = result;
      }
   }

   if (DoCheck) { // report results.
      if (err != 0){
         printf("Error, %d elements do not match!\n", err);
      } else {
         printf("Success! All elements match CPU result.\n");
      }
   }
}

// ------------
// main program
// ------------
int main(void)
{
  int *h_in, *h_out;
  int *d_in, *d_out;
  long size = N * sizeof(int);
  int i;

  // allocate host memory
  h_in = new int[N];
  h_out = new int[N];

  getDeviceProperties();

  // initialize vector
  for (i=0; i<N; i++){
    //    h_in[i] = i+1;
    h_in[i] = 1;
  }

  // allocate device memory
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  cudaErrorCheck();

  // copy input data to device
  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  cudaErrorCheck();

  // Apply stencil by launching a sufficient number of blocks
  printf("\n---------------------------\n");
  printf("Launching 1D stencil kernel\n");
  printf("---------------------------\n");
  printf("Vector length     = %ld (%ld MB)\n",N,N*4/1024/1024);
  printf("Stencil radius    = %d\n",RADIUS);

  //----------------------------------------------------------
  // CODE TO RUN AND TIME THE STENCIL KERNEL.
  //----------------------------------------------------------
  newline();
  printThreadSizes();
  start_timer();
  stencil_1D<<<gridSize,blockSize,(blockSize + 2*RADIUS) * sizeof(int)>>>(d_in, d_out, N);
  std::cout << "Elapsed time: " << stop_timer() << std::endl;
  // copy results back to host
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
  cudaErrorCheck();
  checkResults(h_in, h_out);
  //----------------------------------------------------------

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);
  cudaErrorCheck();
  //=====================================================
  // Evaluate total time of execution with just the CPU.
  //=====================================================
  newline();
  std::cout << "Running stencil with the CPU.\n";
  start_timer();
  // Use checkResults to time CPU version of the stencil with False flag.
  checkResults(h_in, h_out, False);
  std::cout << "Elapsed time: " << stop_timer() << std::endl;
  //=====================================================

  // deallocate host memory
  free(h_in);
  free(h_out);

  return 0;
}
