#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>
#include <omp.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

int deviceCount;

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void sample_vec_add(int size = 1048576)
{
    printf("Sample\n");

    int n = size;
    
    int nBytes = n*sizeof(int);
    
    float *a, *b;  // host data
    float *c;  // results
    float *test;
    
    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipHostMalloc(&test, nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        test[i] = a[i] + b[i];
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    double time_ = omp_get_wtime();
    hipEventRecord(start);
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

    hipMemcpy(c, c_d, nBytes, hipMemcpyDeviceToHost);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    time_ = omp_get_wtime() - time_;
    time_ *= 1000;
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda time: %f ms\nomp time: %f ms\n", milliseconds, time_);
    
    hipDeviceSynchronize();
    
    for(int i = 0; i < n; i++)
    {
        if(c[i] != test[i])
        {
            printf("INCORRECT\n");
            break;
        }
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(test);
}

void streams_vec_add(int size = 1048576)
{
    printf("Streams\n");
    hipSetDevice(1);
    float* temp;
    hipHostMalloc(&temp, sizeof(float));
    hipFree(temp);
    hipSetDevice(0);

    int n = size;
    int nBytes = n * sizeof(float);

    float *a, *b;
    float *c;
    float *test;
    
    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);
    hipHostMalloc(&test, nBytes);

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x/deviceCount));
    
    for(int i=0;i<n;i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        test[i] = a[i] + b[i];
    }

    omp_set_num_threads(deviceCount);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    double time_ = omp_get_wtime();

    #pragma omp parallel
    {
        int thread_num = omp_get_thread_num();
        hipSetDevice(thread_num);

        float *a_d, *b_d;
        float *c_d;
        
        hipMalloc((void **)&a_d, nBytes / deviceCount);
        hipMalloc((void **)&b_d, nBytes / deviceCount);
        hipMalloc((void **)&c_d, nBytes / deviceCount);

        hipMemcpy(a_d, a + thread_num * n / deviceCount, nBytes / deviceCount, hipMemcpyHostToDevice);
        hipMemcpy(b_d, b + thread_num * n / deviceCount, nBytes / deviceCount, hipMemcpyHostToDevice);

        vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

        hipMemcpy(c + thread_num * n / deviceCount, c_d, nBytes / deviceCount, hipMemcpyDeviceToHost);

        hipFree(a_d);
        hipFree(b_d);
        hipFree(c_d);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    time_ = omp_get_wtime() - time_;
    time_ *= 1000;
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda time: %f ms\nomp time: %f ms\n", milliseconds, time_);

    hipDeviceSynchronize();

    for(int i = 0; i < n; i++)
    {
        if(c[i] != test[i])
        {
            printf("INCORRECT!\n");
            break;
        }
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(test);
}


int main(int argc, char **argv)
{

    hipGetDeviceCount(&deviceCount);
    sample_vec_add(atoi(argv[1]));
    streams_vec_add(atoi(argv[1]));

    return 0;
}
