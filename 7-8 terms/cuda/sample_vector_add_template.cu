#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_samle(int size = 1048576)
{
    printf("infied_sample\n");

    int n = size;

    int nBytes = n * sizeof(float);
    float *a, *b;
    float *c;

    hipMallocManaged(&a, nBytes);
    hipMallocManaged(&b, nBytes);
    hipMallocManaged(&c, nBytes);

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    vectorAddGPU<<<grid, block>>>(a, b, c, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void pinned_samle(int size = 1048576)
{
    printf("pinned_sample\n");

    int n = size;

    int nBytes = n * sizeof(float);
    float *a, *b;
    float *c;

    hipHostMalloc(&a, nBytes);
    hipHostMalloc(&b, nBytes);
    hipHostMalloc(&c, nBytes);

    float *a_d, *b_d;
    float *c_d;

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    // printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d, nBytes);
    hipMalloc((void **)&b_d, nBytes);
    hipMalloc((void **)&c_d, nBytes);

    // printf("Copying to device..\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, nBytes, hipMemcpyHostToDevice);
    
    // printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

void usual_sample(int size = 1048576)
{
    printf("usual sample\n");

    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results

    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    // printf("Allocating device memory on host..\n");

    hipMalloc((void **)&a_d, nBytes);
    hipMalloc((void **)&b_d, nBytes);
    hipMalloc((void **)&c_d, nBytes);
    
    // printf("Copying to device..\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    hipMemcpy(a_d, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, nBytes, hipMemcpyHostToDevice);
    
    // printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a);
    free(b);
    free(c);
}


int main(int argc, char **argv)
{
    usual_sample(atoi(argv[1]));
    pinned_samle(atoi(argv[1]));
    unified_samle(atoi(argv[1]));
    
    return 0;
}
